#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <limits>
#include <cmath>
#include "constants.cuh"

using namespace std;

void TDMAsolver(const float* a, const float* b, const float* c, const float* d, float* x) {
	int n = nx;
	float cc[nx - 1] = {0.0f};
	float dc[nx]     = {0.0f};

	// Forward Pass
	cc[0] = c[0] / b[0];
	dc[0] = d[0] / b[0];

	for (int i = 1; i < n - 1; i++) {
		auto den = 1.0f / (b[i] - a[i] * cc[i - 1]);
		cc[i] = c[i] * den;
		dc[i] = (d[i] - a[i] * dc[i - 1]) * den;
	}
	dc[n - 1] = (d[n - 1] - a[n - 1] * dc[n - 2]) / (b[n - 1] - a[n - 1] * cc[n - 2]);

	// Back sub
	x[n - 1] = dc[n - 1];
	for (int i = n - 1; i >= 0; i--) {
		x[i] = dc[i] - cc[i] * x[i + 1];
	}
}

void run_loop(float* Ez, float* By, float* Jz) {
	float a[nx - 1] = {0.0f};
	float b[nx]     = {0.0f};
	float c[nx - 1] = {0.0f};
	float ez_nhalf[nx]     = {0.0f};
	float ez_nhalf_rhs[nx] = {0.0f};

	// Set arrays
	float coeff = 1.0f / (8.0f * eps0 * mu0) * (dt / dx) * (dt / dx);
	for (int i = 0; i < nx - 1; i++) {
		a[i] = -1 * coeff;
		c[i] = -1 * coeff;
		b[i] = 0.5f - coeff * (-2.0f);
	}
	b[nx - 1] = 0.5f - coeff * (-2.0f);

	// Begin time loop
	for (int q = 0; q < nt; q++) {
		// Source
		float t = (float(q) + 0.5f) * dt;
		Jz[0] = 9.89399f * sin(2.0f * pi * freq * t) * exp(-1 * pow(t - t0, n0) / (2.0f * pow(sig0, n0)));

		// Implicit update
		for (int n = 1; n < nx; n++) {
			ez_nhalf_rhs[n] = Ez[n] + (1.0f / eps0) * (0.5f * dt / dx) * (By[n] - By[n-1]) + (0.5f * dt / eps0) * Jz[n-1];
		}
		TDMAsolver(a, b, c, ez_nhalf_rhs, ez_nhalf);

		// Explicit update
		for (int n = 0; n < nx - 1; n++) {
			Ez[n] = ez_nhalf[n] - Ez[n];
			By[n] = By[n] + (1.0f / mu0) * (0.5f * dt / dx) * (ez_nhalf[n + 1] - ez_nhalf[n]);
		}

		// Implicit update (again)
		// Nothing to do here (requires more dimensions)

		if (q % 10 == 0) {
			cout << q << "/" << nt << endl;
			ofstream output("outputs/ez_q" + to_string(q) + ".csv");
			output << setprecision(numeric_limits<float>::max_digits10);

			for (int i = 0; i < nx; i++) {
				output << Ez[i] << "\n";
			}
			output.close();
		}
	}
}

int main() {
	float Jz[nx] = {0.0f};
	float Ez[nx] = {0.0f};
	float By[nx] = {0.0f};

	run_loop(Ez, By, Jz);

	return 0;
}
