#include <iostream>

#include "constants.cuh"

class Efield {
public:
	Efield() : Ez(nullptr) {}
	~Efield() = default;

	void create() {
		checkErr(hipMallocManaged(&Ez, nx * sizeof(float)))
		checkErr(hipDeviceSynchronize())
	}

	void destroy() {
		checkErr(hipDeviceSynchronize())
		checkErr(hipFree(Ez))
	}

	void zero() {
		for (int i = 0; i < nx; i++) {
			Ez[i] = 0.0f;
		}
	}

public:
//	float *Ex;
//	float *Ey;
	float *Ez;
};

class Bfield {
public:
	Bfield() : Bx(nullptr) {}
	~Bfield() = default;

	void create() {
		checkErr(hipMallocManaged(&Bx, nx * sizeof(float)))
		checkErr(hipDeviceSynchronize())
	}

	void destroy() {
		checkErr(hipDeviceSynchronize())
		checkErr(hipFree(Bx))
	}

	void zero() {
		for (int i = 0; i < nx; i++) {
			Bx[i] = 0.0f;
		}
	}
public:
	float* Bx;
//	float* By;
//	float* Bz;
};


int main() {

	Efield e;
	e.create();
	e.zero();

	Bfield b;
	b.create();
	b.zero();

	return 0;
}
