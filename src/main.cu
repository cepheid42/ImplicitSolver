#include "hip/hip_runtime.h"
#include "constants.cuh"
#include "e_field.cuh"
#include "b_field.cuh"
#include "sources.cuh"
#include "update_functions.cuh"
#include "tdma.cuh"
#include "file_io.cuh"

using namespace std;

const int step = 10;

void run_loop(Efield& e, Bfield& b, Source& s) {
	Timer update_loop_timer;
	update_loop_timer.start();

	Tridiagonal td_x_half(ny, ddy);
	Tridiagonal td_y_half(nz, ddz);
	Tridiagonal td_z_half(nx, ddx);

	Tridiagonal td_x_one(nz, ddz);
	Tridiagonal td_y_one(nx, ddx);
	Tridiagonal td_z_one(ny, ddy);

	// Begin time loop
	for (int q = 0; q < nt; q++) {
		// Sources
//		inc_ey(e.Ey, q);
//		inc_ez(e.Ez, q);
		auto ind = true_middle();
		auto a = ((float(q) * dt) - t0) / tau;
		s.Jz[ind] = a * exp(-1.0f * (a * a));

		// c1 = dt / (2 * eps0)
		// c2 = dt / (2 * mu0)

		/* N -> N + 1/2 */
		// Implicit update
		implicit_ex_half(e.ex_rhs, e.Ex, b.Bz, s.Jx); // ex = Ex + c1 * ddy * Bz - c1 * Jx
		implicit_ey_half(e.ey_rhs, e.Ey, b.Bx, s.Jy); // ey = Ey + c1 * ddz * Bx - c1 * Jy
		implicit_ez_half(e.ez_rhs, e.Ez, b.By, s.Jz); // ez = Ez + c1 * ddx * By - c1 * Jz

		ddy_solve(td_x_half, e.ex_rhs, e.ex);
		ddz_solve(td_y_half, e.ey_rhs, e.ey);
		ddx_solve(td_z_half, e.ez_rhs, e.ez);


		// Explicit update
		explicit_E(e.Ex, e.ex);
		explicit_E(e.Ey, e.ey);
		explicit_E(e.Ez, e.ez);

		explicit_bx_half(b.Bx, e.ey); // Bx = Bx + c2 * ddz * ey
		explicit_by_half(b.By, e.ez); // By = By + c2 * ddx * ez
		explicit_bz_half(b.Bz, e.ex); // Bz = Bz + c2 * ddy * ex


		/* N + 1/2 -> N + 1 */
		// Implicit update
		implicit_ex_one(e.ex_rhs, e.Ex, b.By); // ex = Ex - c1 * ddz * By
		implicit_ey_one(e.ey_rhs, e.Ey, b.Bz); // ey = Ey - c1 * ddx * Bz
		implicit_ez_one(e.ez_rhs, e.Ez, b.Bx); // ez = Ez - c1 * ddy * Bx

		ddz_solve(td_x_one, e.ex_rhs, e.ex);
		ddx_solve(td_y_one, e.ey_rhs, e.ey);
		ddy_solve(td_z_one, e.ez_rhs, e.ez);

		// Explicit update
		explicit_E(e.Ex, e.ex);
		explicit_E(e.Ey, e.ey);
		explicit_E(e.Ez, e.ez);

		explicit_bx_one(b.Bx, e.ez); // Bx = Bx - c2 * ddy * ez
		explicit_by_one(b.By, e.ex); // By = By - c2 * ddz * ex
		explicit_bz_one(b.Bz, e.ey); // Bz = Bz - c2 * ddx * ey

		if (q % step == 0) {
			cout << q << "/" << nt << ": snapshot taken (" << update_loop_timer.split() << "s)" << endl;
			snapshot(q, e, b, s);
		}
	}
	update_loop_timer.stop();
	cout << "Total loop time: " << setprecision(3) << update_loop_timer.total << "s" << endl;
}

int main() {
	save_params(step);

	Efield e;
	Bfield b;
	Source s;

	run_loop(e, b, s);
	return 0;
}
